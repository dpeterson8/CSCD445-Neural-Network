#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "mnistFileUtils.h"
#include "cpuNetwork.h"
#include "arrayUtils.h"

#include "gpuNetwork.h"

int INPUTSIZE = 2;
int HIDDENSIZE = 2;
int OUTSIZE = 1;

void usage(){
    printf("./project 1\n");
    printf("The only argument will determine whether the code si run using cpu or gpu side.\n");
    printf("For cpu side './project 1' will cause the cpu side to run, anthing else will run gpu\n");
}

int main( int argc, char *argv[] ) {
    
    // float * inputArr = (float *) malloc(sizeof(float) * 784 * 60000);
    // float * correctInput = (float *) malloc(sizeof(float) * 60000);
    // float * correctData = (float *) (malloc(sizeof(float) * 60000 * 10));

    // getMnistTrain(inputArr, correctInput, correctData, 1);

    // __time_t t;

    if(argc != 2) {
        usage();
        exit(1);
    }
    
    srand((unsigned) time(NULL));
    
    float * inputLayer = (float *) malloc(sizeof(float) * INPUTSIZE);
    float * hiddenOneLayer = (float *) malloc(sizeof(float) * HIDDENSIZE);
    float * hiddenTwoLayer = (float *) malloc(sizeof(float) * HIDDENSIZE);
    float * outLayer = (float *) malloc(sizeof(float) * OUTSIZE);

    float * layers = (float *) malloc(sizeof(float) * INPUTSIZE + (HIDDENSIZE * 2) + OUTSIZE);
    float * weights = (float *) malloc(sizeof(float) * (INPUTSIZE * HIDDENSIZE * 2) + (OUTSIZE * HIDDENSIZE));
    float * biases = (float *) malloc(sizeof(float) * (HIDDENSIZE * 2 + OUTSIZE));

    float * hiddenLayerWeights = (float *) malloc(sizeof(float) * OUTSIZE * INPUTSIZE) ;
    float * hiddenTwoLayerWeights = (float *) malloc(sizeof(float) * HIDDENSIZE * 2);
    float * outLayerWights = (float *) malloc(sizeof(float) * OUTSIZE * HIDDENSIZE);

    float * hiddenLayerBias = (float *) malloc(sizeof(float) * HIDDENSIZE);
    float * hiddenTwoLayerBias = (float *) malloc(sizeof(float) * HIDDENSIZE);
    float * outputLayerBias = (float *) malloc(sizeof(float) * OUTSIZE);

    float orInput[48] = { 0.0, 0.0, 1.0, 0.0, 0.0, 1.0, 1.0, 1.0, 0.0, 0.0, 1.0, 0.0, 0.0, 1.0, 1.0, 1.0,
                       0.0, 0.0, 1.0, 0.0, 0.0, 1.0, 1.0, 1.0, 0.0, 0.0, 1.0, 0.0, 0.0, 1.0, 1.0, 1.0,
                       0.0, 0.0, 1.0, 0.0, 0.0, 1.0, 1.0, 1.0, 0.0, 0.0, 1.0, 0.0, 0.0, 1.0, 1.0, 1.0};
    float inputCorrect[24] = {0.0,1.0,1.0,0.0,0.0,1.0,1.0,0.0,
                              0.0,1.0,1.0,0.0,0.0,1.0,1.0,0.0,
                              0.0,1.0,1.0,0.0,0.0,1.0,1.0,0.0}; 


    float lr = 0.4;
    int epochs = 100000;
    static const int amountOfData = 8;



    // iniate hidden layer weights and biases
    iniateWeigts(hiddenLayerWeights, (INPUTSIZE * HIDDENSIZE));
    iniateWeigts(hiddenLayerBias, (HIDDENSIZE));

    // iniate out layer weights and biases
    iniateWeigts(outLayerWights, (HIDDENSIZE * OUTSIZE));
    iniateWeigts(outputLayerBias, (OUTSIZE));

    if(*argv[1] == '1') {
        
        trainNetwork(inputLayer, hiddenOneLayer, hiddenTwoLayer, outLayer, hiddenLayerWeights, hiddenTwoLayerWeights, outLayerWights, hiddenLayerBias,
                    hiddenTwoLayerBias, outputLayerBias, orInput, inputCorrect, amountOfData, INPUTSIZE, HIDDENSIZE, OUTSIZE, epochs, lr);            
    } else {

        float * d_input;
        float * d_correct;
        size_t d_input_size = sizeof(float) * amountOfData;

        float * d_input_layer;
        size_t d_inputLayer_size = sizeof(float) * INPUTSIZE;

        float * d_hLayerOne;
        float * d_hLayerTwo;
        float * d_outLayer;

        float * d_fWeights;
        float * d_outWeights;

        float * d_fBias;
        float * d_outBias;

        float * d_deltaOut;

        float * d_deltaOne; 

        int hiddenLayerSize = sizeof(float) * HIDDENSIZE;
        int outLayerSize = sizeof(float) * OUTSIZE;

        int hiddenWeightSize = sizeof(float) * HIDDENSIZE * INPUTSIZE;

        hipMalloc((void**)&d_input, d_input_size);
        hipMalloc((void**)&d_correct, (sizeof(float) * OUTSIZE * amountOfData));
        hipMalloc((void**)&d_input_layer, d_inputLayer_size * amountOfData);
        hipMalloc((void**)&d_fWeights, hiddenWeightSize);
        hipMalloc((void**)&d_fBias, (sizeof(float) * HIDDENSIZE));
        hipMalloc((void**)&d_hLayerOne, hiddenLayerSize * amountOfData);
        hipMalloc((void**)&d_hLayerTwo, hiddenLayerSize);
        hipMalloc((void**)&d_outWeights, (sizeof(float) * HIDDENSIZE * OUTSIZE));
        hipMalloc((void**)&d_outBias, (sizeof(float) * OUTSIZE));
        hipMalloc((void**)&d_outLayer, outLayerSize * amountOfData);    
        hipMalloc((void**)&d_deltaOut, (sizeof(float) * OUTSIZE) * amountOfData);
        hipMalloc((void**)&d_deltaOne, (sizeof(float) * HIDDENSIZE) * amountOfData);

        hipMemcpy(d_input_layer, orInput, d_inputLayer_size * amountOfData, hipMemcpyHostToDevice);
        hipMemcpy(d_correct, inputCorrect, (sizeof(float) * OUTSIZE * amountOfData), hipMemcpyHostToDevice);
        hipMemcpy(d_fWeights, hiddenLayerWeights, hiddenLayerSize * INPUTSIZE, hipMemcpyHostToDevice);
        hipMemcpy(d_fBias, hiddenLayerBias, hiddenLayerSize, hipMemcpyHostToDevice);
        hipMemcpy(d_outWeights, outLayerWights, (sizeof(float) * HIDDENSIZE * OUTSIZE), hipMemcpyHostToDevice);
        hipMemcpy(d_outBias, outputLayerBias, outLayerSize, hipMemcpyHostToDevice);

        for(int i = 0; i < epochs; i++) {
            shuffle(orInput, inputCorrect, amountOfData, INPUTSIZE);
            hipMemcpy(d_input_layer, orInput, d_inputLayer_size * amountOfData, hipMemcpyHostToDevice);
            hipMemcpy(d_correct, inputCorrect, (sizeof(float) * OUTSIZE * amountOfData), hipMemcpyHostToDevice);
            gpuTrainNetwork<<<1, 1>>>(d_input_layer, d_hLayerOne, d_outLayer, d_fWeights, d_outWeights, d_fBias, d_outBias, d_input, d_correct, amountOfData,INPUTSIZE, HIDDENSIZE, OUTSIZE, epochs, lr);    
            hipDeviceSynchronize();
        }

        hipFree(d_input);
        hipFree(d_correct);
        hipFree(d_input_layer);
        hipFree(d_fWeights);
        hipFree(d_fBias);
        hipFree(d_hLayerOne);
        hipFree(d_hLayerTwo);
        hipFree(d_outWeights);
        hipFree(d_outBias);
        hipFree(d_outLayer);
        hipFree(d_deltaOut);
        hipFree(d_deltaOne);

    }

    free(inputLayer);
    free(hiddenOneLayer);
    free(hiddenTwoLayer);
    free(outLayer);
    free(layers);
    free(weights);
    free(biases);
    free(hiddenLayerWeights);
    free(hiddenTwoLayerWeights);
    free(hiddenLayerBias);
    free(hiddenTwoLayerBias);
    free(outputLayerBias);

}